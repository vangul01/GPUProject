#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <fstream>
#include<sstream>
#include<string.h>
// #include<bits/stdc++.h> 

using namespace std;
#define index(i, j, colNum)  ((i)*(colNum)) + (j)

#include "0header.h"
#include "matrixCalculationGPU.cpp"

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err)); 
        exit(1);
    }
#endif
    return;
}

//for kernel
inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) { 
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
        exit(1);
    }
    // More careful checking. However, this will affect performance. Comment away if needed.
    //err = hipDeviceSynchronize();
    //if(hipSuccess != err) {
    //    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
    //    exit(1);
    //}
#endif
    return;
}

////////////////////////// KERNELS ///////////////////////////

__global__ void simpleMultiplication(float matA[], int width, int colA, float matB[], int colB, float result[]) {
//	int tx = blockIdx.x * blockDim.x + threadIdx.x;
//	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	float tempSum = 0;

	for(int k = 0; k < colA; k++) {
		float elementA = matA[ty * width + k];
		float elementB = matB[k * width + tx];
		tempSum += elementA * elementB; //matA[index(i, k, colA)]*matB[index(k, j, colB)] ;
	}
	result[ty * width + tx] = tempSum; //[index(i,j, colB)] = tempSum;

}

////////////////////////// MAIN ///////////////////////////

int main(int argc, char *argv[]) {
	int rowNum = atoi(argv[1]);
	int colNum = atoi(argv[2]);

	float* X = (float*)malloc(sizeof(float)*rowNum*colNum); //	float X[rowNum*colNum];
	float* transposeX = (float*)malloc(sizeof(float)*rowNum*colNum); //	float transposeX[colNum*rowNum];
	float* Y = (float*)malloc(sizeof(float)*rowNum); //	float Y[rowNum];

	readFile(X, Y, rowNum, colNum);

	for(int i = 0; i < rowNum; i++) {
		for(int j = 0; j < colNum; j++) {
			transposeX[index(j,i, rowNum)] = X[index(i,j, colNum)];	
		}
	}

	N = colNum;
	float* tranXmulxMat = (float*)malloc(sizeof(float)*colNum*colNum); //float tranXmulxMat[colNum*colNum];
	float* inv = (float*)malloc(sizeof(float)*colNum*colNum); //float inv[N*N]; // To store inverse of A[][] 
	float* resultM = (float*)malloc(sizeof(float)*rowNum*colNum); //float resultM[colNum*rowNum];
	float* finalResult = (float*)malloc(sizeof(float)*colNum); //float finalResult[colNum];

/////////////////////////////////////////DEVICE CODE////////////////////////////////////////

	//declare and allocate mem for device vars
	float* d_X; float* d_transposeX; float* d_Y; 
	float* d_tranXmulxMat; float* d_resultM; float* d_finalResult; float* d_inv; 

	//init d_arrays
	CudaSafeCall(hipMalloc((void**)&d_X, sizeof(float)*rowNum*colNum));	
	CudaSafeCall(hipMalloc((void**)&d_Y, sizeof(float)*rowNum));	
	CudaSafeCall(hipMalloc((void**)&d_transposeX, sizeof(float)*rowNum*colNum));	
	
	//transfer data to device
	hipMemcpy(d_X, X, sizeof(int)*rowNum*colNum, hipMemcpyHostToDevice); 
	hipMemcpy(d_transposeX, transposeX, sizeof(int)*rowNum*colNum, hipMemcpyHostToDevice); 
	hipMemcpy(d_Y, Y, sizeof(int)*rowNum, hipMemcpyHostToDevice); 

	//allocate for results
	CudaSafeCall(hipMalloc((void**)&d_tranXmulxMat, sizeof(float)*colNum*colNum));	
	CudaSafeCall(hipMalloc((void**)&d_inv, sizeof(float)*colNum*colNum));	
	CudaSafeCall(hipMalloc((void**)&d_resultM, sizeof(float)*colNum*rowNum));	
	CudaSafeCall(hipMalloc((void**)&d_finalResult, sizeof(float)*colNum));		

	//setup kernel config
	int threadCount = 32;
	int blocks_per_grid = ((rowNum*colNum)/threadCount + 1);
	dim3 dimGrid(blocks_per_grid, 1, 1); 
	dim3 dimBlock(threadCount, 1, 1);

	cout<<"X\'-------->"<<endl;
	//matrixPrint(transposeX, colNum, rowNum);
	cout<<"X ----------->"<<endl;
	//matrixPrint(X, rowNum, colNum);

	//Call kernel
	cout<<"X\'X----------->"<<endl;
//	matMultiplication(transposeX, colNum, rowNum, X, rowNum, colNum, tranXmulxMat);
	simpleMultiplication<<<dimGrid, dimBlock>>>(d_transposeX, colNum, rowNum, d_X, colNum, d_tranXmulxMat); 
	CudaCheckError();
	hipMemcpy(tranXmulxMat, d_tranXmulxMat, size, hipMemcpyDeviceToHost);
	display(tranXmulxMat, colNum, colNum);

	//No kernel
	cout<<"The inverse is --------->"<<endl;
	if (inverse(tranXmulxMat, inv)) {
		display(inv, colNum, colNum); 
		hipMemcpy(d_inv, inv, size, hipMemcpyHostToDevice);
	}

	//Call kernel...fails
	cout<<"(X'X)^-1*X' ->"<<endl;
	matMultiplication(inv, colNum, colNum, transposeX, colNum, rowNum, resultM);
//	matMultiplication<<<dimGrid, dimBlock>>>(d_inv, colNum, colNum, d_transposeX, colNum, rowNum, d_resultM); 
//	CudaCheckError();
//	hipMemcpy(resultM, d_resultM, size, hipMemcpyDeviceToHost);
	display(resultM, colNum, rowNum);

	//Call kernel...fails
	cout<<"final (X'X)^-1*X'Y ->"<<endl;
	matMultiplication(resultM, colNum, rowNum, Y, rowNum, 1, finalResult);
//	matMultiplication<<<dimGrid, dimBlock>>>(d_resultM, colNum, rowNum, d_Y, rowNum, 1, d_finalResult); 
//	CudaCheckError();
//	hipMemcpy(finalResult, d_finalResult, size, hipMemcpyDeviceToHost);
	display(finalResult, colNum, 1);

//	hipMemcpy(inv, d_inv, size, hipMemcpyDeviceToHost);

////////////////////////////////////////////////////////////////////////////////////////

	free(X); free(transposeX); free(Y);
	free(tranXmulxMat); free(inv); free(resultM); free(finalResult);
	cudafree(d_X); cudafree(d_transposeX); cudafree(d_Y);
	cudafree(d_tranXmulxMat); cudafree(d_inv); cudafree(d_resultM); cudafree(d_finalResult);
	
	return 0;
}



/*
	//matMultiplication(transposeX, colNum, rowNum, X, colNum, tranXmulxMat);
	matMultiplication(transposeX, colNum, rowNum, X, rowNum, colNum, tranXmulxMat);
	cout<<"X\'-------->"<<endl;
	display(transposeX, colNum, rowNum);

	cout<<"X ----------->"<<endl;
	display(X, rowNum, colNum);
	
	cout<<"X\'X----------->"<<endl;
	display(tranXmulxMat, colNum, colNum);

	cout<<"The inverse is --------->"<<endl;
	if (inverse(tranXmulxMat, inv)) 
		display(inv, colNum, colNum); 

	//cal (X'X)^-1*X'
	matMultiplication(inv, colNum, colNum, transposeX, colNum, rowNum, resultM);

	cout<<"(X'X)^-1*X' ->"<<endl;
	display(resultM, colNum, rowNum);
	
	//cal (X'X)^-1*X'Y
	matMultiplication(resultM, colNum, rowNum, Y, rowNum, 1, finalResult);
	cout<<"final (X'X)^-1*X'Y ->"<<endl;
	display(finalResult, colNum, 1);
*///////////////////////////////////////////////////////////
//for cudamalloc
/*
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if (hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err)); 
        exit(1);
    }
#endif
    return;
}

//for kernel
inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) { 
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
        exit(1);
    }
    // More careful checking. However, this will affect performance. Comment away if needed.
    //err = hipDeviceSynchronize();
    //if(hipSuccess != err) {
    //    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
    //    exit(1);
    //}
#endif
    return;
}


__global__ void FindPrimes (int* d_numbers, int N) {
	int tx = blockIdx.x * blockDim.x + threadIdx.x;	

	tx = min(tx, ((N+3)/2)); //instead of if (tx <= ((N+1)/2)), thread is less than or equal to floor, doesnt include N+3/2
	for (int i = tx+1; i < N-1; i++) { //check threads in positions tx+1->N-2
		if (d_numbers[tx] != 1) {
			if (d_numbers[i] % d_numbers[tx] == 0) {
				d_numbers[i] = 1;
			}
		}
	}
}

R=2 C=3
X'-------->
0 3 
1 4 
2 5 
X ----------->
0 1 2 
3 4 5 
X'X----------->
9 12 15 
12 17 22 
15 22 29 


R=2 C=3
X'-------->
0 3 	1 4 	2 5 
X ----------->
0 1 2	 3 4 5 
X'X----------->
9 12 15 	12 17 22	 15 22 29 


// X'(0 3) * X(0 3)

for each row of X'
	for each column of X
		(i = 0; i<X'row length)
			add mult between elements

*/

////////////////////////////////////// KERNELS //////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

//matMultiplication(d_transposeX, colNum, rowNum, d_X, rowNum, colNum, d_tranXmulxMat);
	//matMultiplication(inv, colNum, colNum, transposeX, colNum, rowNum, resultM);
	//matMultiplication(resultM, colNum, rowNum, Y, rowNum, 1, finalResult);
/*__global__ */ /*void matMultiplication(float* matA, int rowA, int colA, float* matB, int rowB, int colB, float* result) {
	int tempSum;
//	int tx = blockIdx.x * blockDim.x + threadIdx.x;
//	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	int width = rowA;

	for(int i = 0; i < width; i++) {
		for(int j = 0; j < width; j++) {
			tempSum = 0;
			for(int k = 0; k < colA; k ++ ) {
				tempSum += matA[index(i, k, colA)]*matB[index(k, j, colB)] ;
			}
			result[index(i,j, colB)] = tempSum;
		}
	}
}
*/
/*
void matMultiplication(float* matA, int rowA, int colA, float* matB, int rowB, int colB, float* resultMat) {
	float tempSum;

	for(int i = 0; i < rowA; i++) {
		for(int j = 0; j < colB; j++) {
			tempSum = 0;
			for(int k = 0; k < colA; k ++ ) {
				tempSum += matA[index(i, k, colA)]*matB[index(k, j, colB)] ;
				// cout<<"tempSum ->"<<tempSum<<endl;
			}
			// tempMat[index(i, j, colB)] = tempSum;
			resultMat[index(i,j, colB)] = tempSum;
		}
	}
}
*/
/*
  //compute only non-edge cases and put into d_result 
  //thread is within bounds of playground dimensions 
  if (tx < N*N) 
  { 
    if (tx > 0 && tx < N-1) //threads that are not on the edges
    { 
        for (int i = 1; i < N-1; i++) //i's that are not on the edges
        { 
          d_result[index(tx, i, N)] = (d_temp[index(tx-1, i, N)] + d_temp[index(tx+1, i, N)] 
            + d_temp[index(tx, i-1, N)] + d_temp[index(tx, i+1, N)])/4.0;
        }
    }
  }

*/


////////////////////////////////// MAIN ///////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////
/*
int main(int argc, char *argv[]) {
	srand(time(NULL)); //use rand()%10 for random bt 0-9

	if (argc != 3) {
		cout << "Improper number of arguments. \nArg 1: Number of Rows\nArg 2: Number of Columns" << endl; 
		exit(1);
	}

	//error check this
	string rowInput = argv[(argc - 2)];
	string colInput = argv[(argc - 1)];
	int rowNum = stoi(rowInput);
	int colNum = stoi(colInput);

	cout <<"Rows: " << rowNum << " Columns: " << colNum << endl;
	
	//populate X with random values between 0-9..right now no for stability
	int count = 0;
	float* X = (float*)malloc(sizeof(float)*rowNum*colNum); 
	for(int i = 0; i < rowNum; i++) {
		for(int j = 0; j < colNum; j++) {
			X[index(i, j, colNum)] = count;//(float)(rand()%10);	
			count++;
		}
	}

	//populate Y with random 0 or 1 value
	float* Y = (float*)malloc(sizeof(float)*rowNum); 
	for(int i = 0; i < rowNum; i++) {
		Y[i] = rand()%2;
	}

	//transpose X on X' and create new matrix, could Potentially use GPU
	float* transposeX = (float*)malloc(sizeof(float)*rowNum*colNum); 
	for(int i = 0; i < rowNum; i++) {
		for(int j = 0; j < colNum; j++) {
			transposeX[index(j,i, rowNum)] = X[index(i,j, colNum)];	
		}
	}

	N = colNum;
	float* tranXmulxMat = (float*)malloc(sizeof(float)*colNum*colNum); 
	float* inv = (float*)malloc(sizeof(float)*colNum*colNum); 
	float* resultM = (float*)malloc(sizeof(float)*rowNum*colNum); 
	float* finalResult = (float*)malloc(sizeof(float)*colNum); 

*/
/////////////////////////////////////////DEVICE CODE////////////////////////////////////////
/*
	//declare and allocate mem for device vars
	float* d_X; float* d_transposeX; float* d_Y; 
	float* d_tranXmulxMat; float* d_resultM; float* d_finalResult;
//	float* d_inv; 

	//for already init CPU arrays
	CudaSafeCall(hipMalloc((void**)&d_X, sizeof(float)*rowNum*colNum));	
	CudaSafeCall(hipMalloc((void**)&d_Y, sizeof(float)*rowNum));	
	CudaSafeCall(hipMalloc((void**)&d_transposeX, sizeof(float)*rowNum*colNum));	
	
	//for results
	CudaSafeCall(hipMalloc((void**)&d_tranXmulxMat, sizeof(float)*colNum*colNum));	
//	CudaSafeCall(hipMalloc((void**)&d_inv, sizeof(float)*colNum*colNum));	
	CudaSafeCall(hipMalloc((void**)&d_resultM, sizeof(float)*colNum*rowNum));	
	CudaSafeCall(hipMalloc((void**)&d_finalResult, sizeof(float)*colNum));		

	//transfer data to device
	hipMemcpy(d_X, X, sizeof(int)*rowNum*colNum, hipMemcpyHostToDevice); 
	hipMemcpy(d_transposeX, transposeX, sizeof(int)*rowNum*colNum, hipMemcpyHostToDevice); 
	hipMemcpy(d_Y, Y, sizeof(int)*rowNum, hipMemcpyHostToDevice); 

	//setup kernel config
	int threadCount = 32;
	int blocks_per_grid = ((rowNum*colNum)/threadCount + 1);
	dim3 dimGrid(blocks_per_grid, 1, 1); 
	dim3 dimBlock(threadCount, 1, 1);
*/
/*
	//No kernel
	cout<<"X\'-------->"<<endl;
	matrixPrint(transposeX, colNum, rowNum);
	
	//No kernel	
	cout<<"X ----------->"<<endl;
	matrixPrint(X, rowNum, colNum);

	//Call kernel
	cout<<"X\'X----------->"<<endl;
	matMultiplication(transposeX, colNum, rowNum, X, rowNum, colNum, tranXmulxMat); 
//	matMultiplication<<<dimGrid, dimBlock>>>(d_transposeX, colNum, rowNum, d_X, rowNum, colNum, d_tranXmulxMat); 
//	CudaCheckError();
//	hipMemcpy(tranXmulxMat, d_tranXmulxMat, size, hipMemcpyDeviceToHost);
	display(tranXmulxMat, colNum, colNum);

	//No kernel
	cout<<"The inverse is --------->"<<endl;
	if (inverse(tranXmulxMat, inv)) 
		display(inv, colNum, colNum); 

	//Call kernel...fails
	cout<<"(X'X)^-1*X' ->"<<endl;
	matMultiplication(inv, colNum, colNum, transposeX, colNum, rowNum, resultM);
//	matMultiplication<<<dimGrid, dimBlock>>>(d_inv, colNum, colNum, d_transposeX, colNum, rowNum, d_resultM); 
//	CudaCheckError();
//	hipMemcpy(resultM, d_resultM, size, hipMemcpyDeviceToHost);
	display(resultM, colNum, rowNum);

	//Call kernel...fails
	cout<<"final (X'X)^-1*X'Y ->"<<endl;
	matMultiplication(resultM, colNum, rowNum, Y, rowNum, 1, finalResult);
//	matMultiplication<<<dimGrid, dimBlock>>>(d_resultM, colNum, rowNum, d_Y, rowNum, 1, d_finalResult); 
//	CudaCheckError();
//	hipMemcpy(finalResult, d_finalResult, size, hipMemcpyDeviceToHost);
	display(finalResult, colNum, 1);

//	hipMemcpy(inv, d_inv, size, hipMemcpyDeviceToHost);

}
*/
////////////////////////////////////////////////////////////////////////////////////////////
/*	
	cout<<"X\'X----------->"<<endl;
	matMultiplication(transposeX, colNum, rowNum, X, rowNum, colNum, tranXmulxMat);
	display(tranXmulxMat, colNum, colNum);

	//cal inverse --------------
	cout<<"The inverse is --------->"<<endl;
	if (inverse(tranXmulxMat, inv)) 
		display(inv, colNum, colNum); 
	
	//cal (X'X)^-1*X' --------------
	cout<<"(X'X)^-1*X' ->"<<endl;
	matMultiplication(inv, colNum, colNum, transposeX, colNum, rowNum, resultM);	
	display(resultM, colNum, rowNum);
	
	//cal (X'X)^-1*X'Y --------------
	cout<<"final (X'X)^-1*X'Y ->"<<endl;
	matMultiplication(resultM, colNum, rowNum, Y, rowNum, 1, finalResult);	
	display(finalResult, colNum, 1);

}
*/